#include "hip/hip_runtime.h"
extern "C" {
#include "network.h"
#include "detection_layer.h"
#include "cost_layer.h"
#include "utils.h"
#include "parser.h"
#include "box.h"
#include "image.h"
}

#ifdef OPENCV
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
extern "C" image ipl_to_image(IplImage* src);
extern "C" void convert_coco_detections(float *predictions, int classes, int num, int square, int side, int w, int h, float thresh, float **probs, box *boxes, int only_objectness);
extern "C" void draw_coco(image im, int num, float thresh, box *boxes, float **probs);

static float **probs;
static box *boxes;
static network net;
static image in   ;
static image in_s ;
static image det  ;
static image det_s;
static image disp ;
static cv::VideoCapture cap(0);

void *fetch_in_thread(void *ptr)
{
    cv::Mat frame_m;
    cap >> frame_m;
    IplImage frame = frame_m;
    in = ipl_to_image(&frame);
    rgbgr_image(in);
    in_s = resize_image(in, net.w, net.h);
    return 0;
}

void *detect_in_thread(void *ptr)
{
    float nms = .4;
    float thresh = .2;

    detection_layer l = net.layers[net.n-1];
    float *X = det_s.data;
    float *predictions = network_predict(net, X);
    free_image(det_s);
    convert_coco_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, thresh, probs, boxes, 0);
    if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
    printf("\033[2J");
    printf("\033[1;1H");
    printf("\nObjects:\n\n");
    draw_coco(det, l.side*l.side*l.n, thresh, boxes, probs);
    return 0;
}

extern "C" void demo_coco(char *cfgfile, char *weightfile, float thresh)
{
    printf("YOLO demo\n");
    net = parse_network_cfg(cfgfile);
    if(weightfile){
        load_weights(&net, weightfile);
    }
    set_batch_network(&net, 1);

    srand(2222222);

    if(!cap.isOpened()) error("Couldn't connect to webcam.\n");

    detection_layer l = net.layers[net.n-1];
    int j;

    boxes = (box *)calloc(l.side*l.side*l.n, sizeof(box));
    probs = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
    for(j = 0; j < l.side*l.side*l.n; ++j) probs[j] = (float *)calloc(l.classes, sizeof(float *));

    pthread_t fetch_thread;
    pthread_t detect_thread;

    fetch_in_thread(0);
    det = in;
    det_s = in_s;

    fetch_in_thread(0);
    detect_in_thread(0);
    disp = det;
    det = in;
    det_s = in_s;

    while(1){
        if(pthread_create(&fetch_thread, 0, fetch_in_thread, 0)) error("Thread creation failed");
        if(pthread_create(&detect_thread, 0, detect_in_thread, 0)) error("Thread creation failed");
        show_image(disp, "YOLO");
        free_image(disp);
        cvWaitKey(1);
        pthread_join(fetch_thread, 0);
        pthread_join(detect_thread, 0);

        disp  = det;
        det   = in;
        det_s = in_s;
    }
}
#else
extern "C" void demo_coco(char *cfgfile, char *weightfile, float thresh){
    fprintf(stderr, "YOLO-COCO demo needs OpenCV for webcam images.\n");
}
#endif

